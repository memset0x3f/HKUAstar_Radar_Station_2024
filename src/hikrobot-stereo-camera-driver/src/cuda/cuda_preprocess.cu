#include "hip/hip_runtime.h"
#include <cstdint>
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <chrono>
#include <ros/ros.h>
#include "cuda_preprocess.hpp"


__global__ void resize(const uchar* srcData, const int srcH, const int srcW, uchar* tgtData, const int tgtH, const int tgtW)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = ix + iy * tgtW;
    int idx3 = idx * 3;

    float scaleY = (float)tgtH / (float)srcH;
    float scaleX = (float)tgtW / (float)srcW;

    // (ix,iy)为目标图像坐标
    // (before_x,before_y)原图坐标
    float beforeX = float(ix + 0.5) / scaleX - 0.5;
    float beforeY = float(iy + 0.5) / scaleY - 0.5;
    // 原图像坐标四个相邻点
    // 获得变换前最近的四个顶点,取整
    int topY = static_cast<int>(beforeY);
    int bottomY = topY + 1;
    int leftX = static_cast<int>(beforeX);
    int rightX = leftX + 1;
    //计算变换前坐标的小数部分
    float u = beforeX - leftX;
    float v = beforeY - topY;

    if (ix < tgtW && iy < tgtH)
    {
        // 如果计算的原始图像的像素大于真实原始图像尺寸
        if (topY >= srcH - 1 && leftX >= srcW - 1)  //右下角
        {
            for (int k = 0; k < 3; k++)
            {
                tgtData[idx3 + k] = (1. - u) * (1. - v) * srcData[(leftX + topY * srcW) * 3 + k];
            }
        }
        else if (topY >= srcH - 1)  // 最后一行
        {
            for (int k = 0; k < 3; k++)
            {
                tgtData[idx3 + k]
                = (1. - u) * (1. - v) * srcData[(leftX + topY * srcW) * 3 + k]
                + (u) * (1. - v) * srcData[(rightX + topY * srcW) * 3 + k];
            }
        }
        else if (leftX >= srcW - 1)  // 最后一列
        {
            for (int k = 0; k < 3; k++)
            {
                tgtData[idx3 + k]
                = (1. - u) * (1. - v) * srcData[(leftX + topY * srcW) * 3 + k]
                + (1. - u) * (v) * srcData[(leftX + bottomY * srcW) * 3 + k];
            }
        }
        else  // 非最后一行或最后一列情况
        {
            for (int k = 0; k < 3; k++)
            {
                tgtData[idx3 + k]
                = (1. - u) * (1. - v) * srcData[(leftX + topY * srcW) * 3 + k]
                + (u) * (1. - v) * srcData[(rightX + topY * srcW) * 3 + k]
                + (1. - u) * (v) * srcData[(leftX + bottomY * srcW) * 3 + k]
                + u * v * srcData[(rightX + bottomY * srcW) * 3 + k];
            }
        }
    }
}

__global__ void letterBox(const uchar* srcData, const int srcH, const int srcW, uchar* tgtData, 
    const int tgtH, const int tgtW, const int rszH, const int rszW, const int startY, const int startX)
{
    int ix = threadIdx.x + blockDim.x * blockIdx.x;
    int iy = threadIdx.y + blockDim.y * blockIdx.y;
    int idx = ix + iy * tgtW;
    int idx3 = idx * 3;

    if ( ix > tgtW || iy > tgtH ) return;  // thread out of target range
    // gray region on target image
    if ( iy < startY || iy > (startY + rszH - 1) ) {
        tgtData[idx3] = 128;
        tgtData[idx3 + 1] = 128;
        tgtData[idx3 + 2] = 128;
        return;
    }
    if ( ix < startX || ix > (startX + rszW - 1) ){
        tgtData[idx3] = 128;
        tgtData[idx3 + 1] = 128;
        tgtData[idx3 + 2] = 128;
        return;
    }

    float scaleY = (float)rszH / (float)srcH;
    float scaleX = (float)rszW / (float)srcW;

    // (ix,iy)为目标图像坐标
    // (before_x,before_y)原图坐标
    float beforeX = float(ix - startX + 0.5) / scaleX - 0.5;
    float beforeY = float(iy - startY + 0.5) / scaleY - 0.5;
    // 原图像坐标四个相邻点
    // 获得变换前最近的四个顶点,取整
    int topY = static_cast<int>(beforeY);
    int bottomY = topY + 1;
    int leftX = static_cast<int>(beforeX);
    int rightX = leftX + 1;
    //计算变换前坐标的小数部分
    float u = beforeX - leftX;
    float v = beforeY - topY;

    if (topY >= srcH - 1 && leftX >= srcW - 1)  //右下角
    {
        for (int k = 0; k < 3; k++)
        {
            tgtData[idx3 + k] = (1. - u) * (1. - v) * srcData[(leftX + topY * srcW) * 3 + k];
        }
    }
    else if (topY >= srcH - 1)  // 最后一行
    {
        for (int k = 0; k < 3; k++)
        {
            tgtData[idx3 + k]
            = (1. - u) * (1. - v) * srcData[(leftX + topY * srcW) * 3 + k]
            + (u) * (1. - v) * srcData[(rightX + topY * srcW) * 3 + k];
        }
    }
    else if (leftX >= srcW - 1)  // 最后一列
    {
        for (int k = 0; k < 3; k++)
        {
            tgtData[idx3 + k]
            = (1. - u) * (1. - v) * srcData[(leftX + topY * srcW) * 3 + k]
            + (1. - u) * (v) * srcData[(leftX + bottomY * srcW) * 3 + k];
        }
    }
    else  // 非最后一行或最后一列情况
    {
        for (int k = 0; k < 3; k++)
        {
            tgtData[idx3 + k]
            = (1. - u) * (1. - v) * srcData[(leftX + topY * srcW) * 3 + k]
            + (u) * (1. - v) * srcData[(rightX + topY * srcW) * 3 + k]
            + (1. - u) * (v) * srcData[(leftX + bottomY * srcW) * 3 + k]
            + u * v * srcData[(rightX + bottomY * srcW) * 3 + k];
        }
    }
}


__global__ void process(const uchar* srcData, float* tgtData, const int h, const int w, bool doNormalize=true)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = ix + iy * w;
    int idx3 = idx * 3;

    if (ix < w && iy < h)
    {
        tgtData[idx] = (float)srcData[idx3 + 2];  // R pixel
        tgtData[idx + h * w] = (float)srcData[idx3 + 1];  // G pixel
        tgtData[idx + h * w * 2] = (float)srcData[idx3];  // B pixel
        if(doNormalize)
        {
            tgtData[idx] /= 255;
            tgtData[idx + h * w] /= 255;
            tgtData[idx + h * w * 2] /= 255;
        }
    }
}


void cuda_preprocess(hipStream_t &stream, const cv::Mat& srcImg, float* dstDevData, const int dstHeight, const int dstWidth, bool doNormalize)
{
    int srcHeight = srcImg.rows;
    int srcWidth = srcImg.cols;
    int srcElements = srcHeight * srcWidth * 3;
    int dstElements = dstHeight * dstWidth * 3;
    int letterBoxH, letterBoxW, startX, startY;
    float scale = std::min(static_cast<float>(dstWidth) / srcWidth, static_cast<float>(dstHeight) / srcHeight);
    letterBoxH = static_cast<int>(srcHeight * scale);
    letterBoxW = static_cast<int>(srcWidth * scale);
    startY = (dstHeight - letterBoxH) / 2;
    startX = (dstWidth - letterBoxW) / 2;

    // source images data on device
    uchar *srcDevData, *midDevData;
    hipMallocAsync((void**)&midDevData, sizeof(uchar) * dstElements, stream);
    hipHostRegister(srcImg.data, sizeof(uchar) * srcElements, hipHostRegisterMapped);
    hipHostGetDevicePointer(&srcDevData, srcImg.data, 0);


    
    // hipMemcpyAsync(srcDevData, srcImg.data, sizeof(uchar) * srcElements, hipMemcpyHostToDevice, stream);


    dim3 blockSize(32, 32);
    dim3 gridSize((dstWidth + blockSize.x - 1) / blockSize.x, (dstHeight + blockSize.y - 1) / blockSize.y);
    // resize<<<gridSize, blockSize, 0, stream>>>(srcDevData, srcHeight, srcWidth, midDevData, dstHeight, dstWidth);
    letterBox<<<gridSize, blockSize, 0, stream>>>(srcDevData, srcHeight, srcWidth, midDevData, dstHeight, dstWidth, letterBoxH, letterBoxW, startY, startX);
    // hwc to chw / bgr to rgb / normalize
    process<<<gridSize, blockSize, 0, stream>>>(midDevData, dstDevData, dstHeight, dstWidth, doNormalize);
    // hipMemcpy(dstData, dstDevData, sizeof(float) * dstElements, hipMemcpyDeviceToHost);
    // printf("=>need time:%.2f ms\n", ((double)cv::getTickCount() - gtct_time) / ((double)cv::getTickFrequency()) * 1000);
    hipFreeAsync(srcDevData, stream);
    hipFreeAsync(midDevData, stream);

    hipHostUnregister(srcImg.data);

}
